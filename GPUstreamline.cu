#include "hip/hip_runtime.h"
//#include <thrust/host_vector.h>
//#include <thrust/device_vector.h>
#include <stdlib.h>
#include <math.h>
//#include <thrust/copy.h>
//#include <thrust/fill.h>
//#include <thrust/sequence.h>
#include <hip/hip_runtime.h>
//#include <cublacs.h>
#include <iostream>
#include <vector>
#include <stdlib.h>
#include <math.h>
#include <vtkDataArray.h>
#include <vtkFloatArray.h>
#include <vtkPointData.h>
#include "vtkImageData.h"
#include "vtkObjectFactory.h"
#include "vtkStreamingDemandDrivenPipeline.h"
#include "vtkStructuredPointsReader.h"
#include "vtkInformationVector.h"
#include "vtkInformation.h"
#include "vtkDataObject.h"
#include "vtkSmartPointer.h"
#include <vtkFieldData.h>
#include <vtkStructuredPoints.h>
#include <vtkDataSetAttributes.h>
#include <vtkImageReader.h>
#include <iostream>
#include "vtkConfigure.h"
#include <vtkStructuredPointsWriter.h>
using namespace std;



__device__ float* PointVelocityInterpolation(float* point,float* InputData,float* InputSpace, int* InputDims);

__global__ void GetStreamLineLastPoints(float* seeds,float* InputData,float* InputSpace,
	int* InputDims,int step,float timestep,float* StreamLineLastPoints);


__device__ float* PointVelocityInterpolation(float* point,float* InputData,float* InputSpace, int* InputDims)
{
	float subData[24];//8 velocities and 3 values for every velocity
	float dx=InputSpace[0];
	float dy=InputSpace[1];
	float dt=InputSpace[2];
    int InPX=floor(point[0]/dx);//x left of the initialpoint
    int InPY=floor(point[1]/dy);// Y below of the initialpoint
    int InPt=floor(point[2]/dt);//the time below the initialpoint
    int xp=InputDims[0];
    int yp=InputDims[1];
    int tp=InputDims[2];
	int PXY=xp*yp; //how many points in one slice
	float PointVelocityFinal[3];
	float PointVelocity0[3];
	float PointVelocity1[3];
	//cout<<"for point ("<<point[0]<<","<<point[1]<<","<<point[2]<<")"<<endl;
	if(InPX<0||InPX>(xp-1)||InPY<0||InPY>(yp-1)||InPt<0||InPt>2)//only three slice
	{
		//cout<<"the point is out of the area now, Streamline ends"<<endl;
		PointVelocityFinal[2]=1;
	}
	else
	{
		for(int i=0;i<8;i++)
		{
			subData[i*3]=InputData[3*(InPt*PXY+InPY*yp+InPX+i)];
			subData[i*3+1]=InputData[3*(InPt*PXY+InPY*yp+InPX+i)+1];
			subData[i*3+2]=InputData[3*(InPt*PXY+InPY*yp+InPX+i)+2];
			//cout<<"subVelocity of "<<i<<" ("<<subData[i*3]<<","<<subData[i*3+1]<<","<<subData[i*3+2]<<")"<<endl;
		}
	    float d0=point[0]-dx*InPX;//the distance from the first point in x axic
	    float d1=point[1]-dy*InPY;//the distance from the first point in y axic
		PointVelocity0[0]=(1-d1/dy)*((1-d0/dx)*subData[0]+(d0/dx)*subData[3])+(d1/dy)*((1-d0/dx)*subData[6]+(d0/dx)*subData[9]);
	    PointVelocity0[1]=(1-d1/dy)*((1-d0/dx)*subData[1]+(d0/dx)*subData[4])+(d1/dy)*((1-d0/dx)*subData[7]+(d0/dx)*subData[10]);
	    PointVelocity1[0]=(1-d1/dy)*((1-d0/dx)*subData[12]+(d0/dx)*subData[15])+(d1/dy)*((1-d0/dx)*subData[18]+(d0/dx)*subData[21]);
	    PointVelocity1[1]=(1-d1/dy)*((1-d0/dx)*subData[13]+(d0/dx)*subData[16])+(d1/dy)*((1-d0/dx)*subData[19]+(d0/dx)*subData[22]);
         //cout<<"InitialVelocity "<<InitialsubData[0]<<" "<<InitialsubData[1]<<endl;
		float d2=point[2]-dt*InPt;
		PointVelocityFinal[0]=(1-d2/dt)*PointVelocity0[0]+(d2/dt)*PointVelocity1[0];
		PointVelocityFinal[1]=(1-d2/dt)*PointVelocity0[1]+(d2/dt)*PointVelocity1[1];
		PointVelocityFinal[2]=0;
		//cout<<"interpolation Velocity of "<<" ("<<PointVelocityFinal[0]<<","<<PointVelocityFinal[1]<<","<<PointVelocityFinal[2]<<")"<<endl;

	}
	return PointVelocityFinal;
}

__global__ void GetStreamLinePoints(float* seeds,float* InputData,float* InputSpace,
	int* InputDims,int step,float timestep,float* StreamLineLastPoints, int xs, int ys)
{
	int index = blockDim.x * blockIdx.x + threadIdx.x;
	int N=(xs+1)*(ys+1);
    if(index<N)
    {
    	
    	float OneSeed[3];
    	OneSeed[0]=seeds[index*3];
    	OneSeed[1]=seeds[index*3+1];
    	OneSeed[2]=seeds[index*3+2];
        //cout<<"point "<<index<<"("<<OneSeed[0]<<","<<OneSeed[1]<<","<<OneSeed[2]<<")"<<endl;
    	float* InitialVelocity;
		for (int i=0; i<step;i++)
		{
			InitialVelocity=PointVelocityInterpolation(OneSeed,InputData,InputSpace,InputDims);
			if (InitialVelocity[0]==0&&InitialVelocity[1]==0)
		    {
			    //std::cout<<"seed "<<" at step "<<i<<" ,the velocity is 0, streamline finished"<<endl;
			    break;
		    }
		    if(InitialVelocity[2]==1)
		    	{//std::cout<<"point "<<"at step "<<i<<endl;
		            break;
		        }
		    else
		    {
		        float K1[3];
		        K1[0]=InitialVelocity[0];
		        K1[1]=InitialVelocity[1];
		        K1[2]=InitialVelocity[2];
		        float K2Point[3];
		        K2Point[0]=OneSeed[0]+timestep*K1[0]*0.5;
		        K2Point[1]=OneSeed[1]+timestep*K1[1]*0.5;
		        K2Point[2]=OneSeed[2];
		        float* K2;
		        K2=PointVelocityInterpolation(K2Point,InputData,InputSpace,InputDims);
		        if(K2[2]==1)
		        {//cout<<"point "<<index<<" at step "<<i<<endl;
		            break;
		        }
			    else
			    {
				    float K3Point[3];
		            K3Point[0]=OneSeed[0]+timestep*K2[0]*0.5;
		            K3Point[1]=OneSeed[1]+timestep*K2[1]*0.5;
		            K3Point[2]=OneSeed[2];
		            float* K3=PointVelocityInterpolation(K3Point,InputData,InputSpace,InputDims);;
		           if(K3[2]==1)
		           {//cout<<"point "<<"at step "<<i<<endl;
		                break;
		            }
				   else
				   {
				        float K4Point[3];
		                K4Point[0]=OneSeed[0]+timestep*K3[0];
		                K4Point[1]=OneSeed[1]+timestep*K3[1];
	                    K4Point[2]=OneSeed[2];
		                float* K4;
		                K4=PointVelocityInterpolation(K4Point,InputData,InputSpace,InputDims);;
		                if(K4[2]==1)
		                {//cout<<"point "<<"at step "<<i<<endl;
		                    break;
		                }
		                else
		               {
		                    OneSeed[0]=OneSeed[0]+timestep*(K1[0]+2*K2[0]+2*K3[0]+K4[0])/6;
		                    OneSeed[1]=OneSeed[1]+timestep*(K1[1]+2*K2[1]+2*K3[1]+K4[1])/6;
		                    OneSeed[2]=OneSeed[2];
					   }
		            
		           }
		       }
		   }
		   	   // cout<<"point "<<index<<" "<<i<<"("<<OneSeed[0]<<","<<OneSeed[1]<<","<<OneSeed[2]<<")"<<endl;
	    }

	    StreamLineLastPoints[index*3]=OneSeed[0];
        StreamLineLastPoints[index*3+1]=OneSeed[1];
        StreamLineLastPoints[index*3+2]=OneSeed[2];
    }
    index = blockDim.x * blockIdx.x +index;
}



int main(void)
{
	// Read the file
	vtkSmartPointer<vtkStructuredPointsReader> reader = vtkSmartPointer<vtkStructuredPointsReader>::New();
    //reader->SetFileName("createlessdata.vtk");
    reader->SetFileName("velotimeWeight.vtk");
    //reader->SetFileName("velotimeWeight1.vtk");
    reader->Update();
    vtkSmartPointer<vtkStructuredPoints> Data= vtkSmartPointer<vtkStructuredPoints>::New();
    Data = reader->GetOutput();  
    int* dims = Data->GetDimensions();
    double* space1 = Data->GetSpacing();
    float space[3];
    space[0]=(float)space1[0];
    space[1]=(float)space1[1];
    space[2]=(float)space1[2];
    std::cout<<"here is the space  "<<space[0]<<" "<<space[1]<<" "<<space[2]<<std::endl;
    std::cout<<"here is the dims  "<<dims[0]<<" "<<dims[1]<<" "<<dims[2]<<std::endl;
    vtkPointData* points;
    points = Data->GetPointData();
    vtkDataArray* OriginalData;
    OriginalData = points->GetArray(0);
    int NumberofTuple=OriginalData->GetNumberOfTuples();
	float startX=17;
	float endX=20;
    float startY=17;
	float endY=20;
	float startT=100;
	float endT=200;
	int intstartX=floor(startX);
	int intendX=ceil(endX);
	int intstartY=floor(startY);
	int intendY=ceil(endY);
	int intstartT=floor(startT);
	int intendT=ceil(endT);		
	int StepsPerCell=1;
	int Slice=1;
	int Ndims[3]={(intendX-intstartX)*StepsPerCell+1,(intendY-intstartY)*StepsPerCell+1,Slice};

    //device_vector<float> d_InputData(dims[0]*dims[1]*3*3);
    float InputData[dims[0]*dims[1]*3*3];
    int intT=intstartT;
    for(int i=0;i<3*dims[0]*dims[1];i++)//for streamline three layer velocity
	{
		InputData[3*i]=OriginalData->GetTuple(i+intT*dims[0]*dims[1])[0];
		InputData[3*i+1]=OriginalData->GetTuple(i+intT*dims[0]*dims[1])[1];
		InputData[3*i+2]=OriginalData->GetTuple(i+intT*dims[0]*dims[1])[2];	
		//cout<<"input velocity ("<<InputData[3*i]<<","<<InputData[3*i+1]<<","<<InputData[3*i+2]<<")"<<endl;
	} 
    //get seeds
	int xs=floor((endX-startX)*StepsPerCell);//how many space in x
	int ys=floor((endY-startY)*StepsPerCell);
	cout<<"xs and ys :"<<xs<<", "<<ys<<endl;
	//device_vector<float> d_seeds(xs*ys*3);//seeds of streamline
	float seeds[(1+xs)*(1+ys)*3];
	for(int j=0; j<=ys;j++)
	{
		for(int i=0;i<=xs;i++)
		{
			seeds[(j*(xs+1)+i)*3]=(i/StepsPerCell+startX)*space[0];
			seeds[(j*(xs+1)+i)*3+1]=(j/StepsPerCell+startY)*space[1];
		    seeds[(j*(xs+1)+i)*3+2]=(startT-intT)*space[2];
			cout<<"test initial point "<<seeds[(j*(xs+1)+i)*3]<<" "<<seeds[(j*(xs+1)+i)*3+1]<<" "<<seeds[(j*(xs+1)+i)*3+2]<<endl;
		}	
	}

	float InputSpace[3];
	InputSpace[0]=space[0];
	InputSpace[1]=space[1];
	InputSpace[2]=space[2];

    int InputDims[3];
    InputDims[0]=dims[0];
    InputDims[1]=dims[1];
    InputDims[2]=dims[2];

	float StreamLastPoints[(1+xs)*(ys+1)*3];

    //float* ptrInputSpace=raw_pointer_cast(&d_InputSpace[0]);
    //int* ptrInputSDims=raw_pointer_cast(&d_InputDims[0]);
	//float* ptrInputData=raw_pointer_cast(&d_InputData[0]);
	//float* ptrseeds=raw_pointer_cast(&d_seeds[0]);
	//float* ptrStreamLastPoints=raw_pointer_cast(&d_StreamLastPoints);	

	int cellstep=4;
	float timestep=0.001/(0.04*cellstep);//because every cell 4 steps 0.001/(0.04*4)
	cout<<"xs and ys :"<<xs<<", "<<ys<<endl;
	cout<<"timestep : "<<timestep<<endl;
    int Stoptime=intendT-intstartT-1;
	int step=Stoptime*cellstep;
	cout<<"step : "<<step<<endl;
	int * d_step;
	int * d_xs;
	int * d_ys;
    float * d_timestep;
    int* d_InputDims;
    float* d_InputSpace;
    float* d_InputData;
    float* d_seeds;
    float* d_StreamLastPoints;

    hipMalloc((void**)&d_timestep, sizeof(float));
    hipMalloc((void**)&d_step, sizeof(int));
    hipMalloc((void**)&d_xs, sizeof(int));
    hipMalloc((void**)&d_ys, sizeof(int));
    hipMalloc((void**)&d_InputDims, sizeof(int)*3);
    hipMalloc((void**)&d_InputSpace, sizeof(float)*3);
    hipMalloc((void**)&d_InputData, sizeof(float)*dims[0]*dims[1]*3*3);
    hipMalloc((void**)&d_seeds, sizeof(float)*xs*ys*3);
    hipMalloc((void**)&d_StreamLastPoints, sizeof(float)*xs*ys*3);

    hipMemcpy(d_step, &step, sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_xs, &xs, sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_ys, &ys, sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_timestep, &timestep, sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_InputDims, &InputDims,sizeof(int)*3, hipMemcpyHostToDevice);
    hipMemcpy(d_InputSpace, &InputSpace,sizeof(float)*3, hipMemcpyHostToDevice);
    hipMemcpy(d_InputData, &InputData, sizeof(float)*dims[0]*dims[1]*3*3, hipMemcpyHostToDevice);
    hipMemcpy(d_seeds, &seeds, sizeof(float)*xs*ys*3, hipMemcpyHostToDevice);
    hipMemcpy(d_StreamLastPoints, &StreamLastPoints, sizeof(float)*xs*ys*3, hipMemcpyHostToDevice);

	GetStreamLinePoints<<<1,1>>>(d_seeds,d_InputData,d_InputSpace,d_InputDims,step,timestep,
		d_StreamLastPoints,xs,ys);
	//host_vector StreamLineLastPointsResult(d_seeds.size());
	float StreamLineLastPointsResult[xs*ys*3];
	hipMemcpy(StreamLineLastPointsResult,d_StreamLastPoints,sizeof(float)*xs*ys*3,hipMemcpyHostToDevice);
	for(int i=0;i<(xs+1)*(ys+1);i++)
	{
		cout<<"the "<<i<<"th point and the last point is ("<<StreamLineLastPointsResult[3*i]<<", "
		<<StreamLineLastPointsResult[3*i+1]<<", "<<StreamLineLastPointsResult[3*i+2]<<")"<<endl;
	}
	hipFree(d_step);
	hipFree(d_xs);
	hipFree(d_ys);
	hipFree(d_timestep);
	hipFree(d_StreamLastPoints);
	hipFree(d_InputDims);
	hipFree(d_InputSpace);
	hipFree(d_InputData);
	hipFree(d_seeds);
    //StreamLineLastPoints=d_StreamLineLastPoints;
    return 0;

}


